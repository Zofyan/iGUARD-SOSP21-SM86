#include "hip/hip_runtime.h"
/********************************************************************************************
 * Copyright (c) 2021 Indian Institute of Science
 * All rights reserved.
 *
 * Developed by:    Aditya K Kamath
 *                  Computer Systems Lab
 *                  Indian Institute of Science
 *                  https://csl.csa.iisc.ac.in/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of Computer Systems Lab, Indian Institute of Science, 
 *        nor the names of its contributors may be used to endorse or promote products 
 *        derived from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 ********************************************************************************************/
#include "nvbit_tool.h"
#include "nvbit.h"

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string>
#include <unordered_map>
#include <unordered_set>
#include <tuple>
#include <chrono>

//#define DEBUG_OUT
/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "helper.h"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;
std::chrono::time_point<std::chrono::high_resolution_clock> start;
std::chrono::time_point<std::chrono::high_resolution_clock> start_kernel;
std::chrono::time_point<std::chrono::high_resolution_clock> start_time;

double init_time = 0;
double instru_time = 0;
double thread_time = 0;
double kernel_time = 0;
double setup_time = 0;

size_t initial_used = 0;
size_t maxGPUMem = 0;
size_t maxCPUMem = 0;
size_t userGPUMem = 0;
uint64_t prev_used = 0;

/* Counters for race detection */
__managed__ void *counters[TOTAL_CTRS];
__managed__ int parameters[TOTAL_PARAMS];
__managed__ uint64_t mdArrayLen = 100;

/* Details of allocated memory */
__managed__ uint64_t *metadata[TOTAL_MD] = {NULL, NULL};
__managed__ uint64_t addrRangeStart = NULL;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;
bool started = false;

uint64_t dataSize = 100;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;
int turned_off = 0;
int granularity = 4;
int check_locking = 1;
int check_its = 1;
//int lock_granularity = 0;
int race_exit = 0;
int md_scale = 1;
int timeout = 0;
int managed = 1;
int debug_out = 1;

/* opcode to id map and reverse map  */
std::unordered_map<std::string, int> opcode_to_id_map;
std::unordered_map<int, std::string> id_to_opcode_map;

std::unordered_map<hipDeviceptr_t, size_t> ptrSizes;

void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool (def = 0)");
    GET_VAR_INT(turned_off, "TOOL_OFF", 0, "Do not instrument/detect (def = 0)");
    GET_VAR_INT(granularity, "BYTE_GRAN", 4, "Granularity of detection in bytes (def = 4)");
    GET_VAR_INT(check_locking, "CHECK_LOCKS", 1, "Whether to do lockset detection (def = 1)");
    //GET_VAR_INT(lock_granularity, "LOCK_GRAN", 0, "Granularity of lock tracking (0 = warp, 1 = thread; def = 0)");
    GET_VAR_INT(check_its, "CHECK_ITS", 1, "Whether to consider ITS when checking (def = 1)");
    GET_VAR_INT(race_exit, "EXIT", 0, "Quit on encountering error (def = 0)");
    GET_VAR_INT(md_scale, "MD_SCALE", 1, "Factor by which to scale down metadata (def = 1)");
    GET_VAR_INT(timeout, "TIMEOUT", 0, "Time in seconds after which to quit detection (0 = never; def = 0)");
    GET_VAR_INT(managed, "MANAGED", 1, "Force detector to use hipMallocManaged for metadata (def = 1)");
    GET_VAR_INT(debug_out, "DEBUG", 0, "Output debug info (def = 0)");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}
/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    if(debug_out)
        start_time = std::chrono::high_resolution_clock::now();
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
        if (verbose) {
            printf("Inspecting function %s at address 0x%lx\n",
                   nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
        }

        uint32_t cnt = 0;
        /* iterate on all the static instructions in the function */
        for (auto instr : instrs) {
            if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
                    (instr->getMemOpType() == Instr::memOpType::NONE && 
                    !isBarrier(instr) && !isFence(instr) && !(isWarpBar(instr) && check_its))) {
                cnt++;
                continue;
            }
            
            cnt++;
            if (verbose) {
                instr->printDecoded();
            }
            
            if(isBarrier(instr)) {
                /* insert call to the instrumentation function with its
                 * arguments */
                nvbit_insert_call(instr, "instrument_barrier", IPOINT_AFTER);
                /* predicate value */
                nvbit_add_call_arg_pred_val(instr);
                nvbit_add_call_arg_const_val64(instr, (uint64_t)&counters[BARRIER]);
                continue;
            }
            
            if(isFence(instr)) {
                /* insert call to the instrumentation function with its
                 * arguments */
                nvbit_insert_call(instr, "instrument_fence", IPOINT_BEFORE);
                /* predicate value */
                nvbit_add_call_arg_pred_val(instr);
                nvbit_add_call_arg_const_val32(instr, getScope(instr));
                nvbit_add_call_arg_const_val64(instr, (uint64_t)&counters[WARP_CTRS]);
                nvbit_add_call_arg_const_val64(instr, (uint64_t)&counters[LOCKS]);
                nvbit_add_call_arg_const_val64(instr, (uint64_t)parameters);
                continue;
            }
            
            if(isWarpBar(instr) && check_its) {
                /* insert call to the instrumentation function with its
                 * arguments */
                nvbit_insert_call(instr, "instrument_warp_bar", IPOINT_BEFORE);
                /* predicate value */
                nvbit_add_call_arg_pred_val(instr);
                nvbit_add_call_arg_const_val64(instr, (uint64_t)&counters[WARP_BAR]);                
                continue;            
            }

            std::string opcode = std::string(nvbit_get_func_name(ctx, f)) + instr->getSass();
            
            if (opcode_to_id_map.find(opcode) ==
                opcode_to_id_map.end()) {
                int opcode_id = opcode_to_id_map.size();
                opcode_to_id_map[opcode] = opcode_id;
                
                char* file_name;
                char* dir_name;
                uint32_t line;
                bool avail = nvbit_get_line_info(ctx, f, instr->getOffset(), &file_name, &dir_name, &line);
                std::string output;
                if(avail)
                    output = std::string(file_name) + " - Kernel " + std::string(nvbit_get_func_name(ctx, f)) + ": Line " + std::to_string(line) + "\t" + instr->getSass();
                else
                    output = std::string(instr->getSass()) + " - Kernel " + std::string(nvbit_get_func_name(ctx, f)) + ": Sass offset " + std::to_string(instr->getOffset());
                id_to_opcode_map[opcode_id] = output;
            }
            
            int opcode_id = opcode_to_id_map[opcode];
            int mref_idx = 0;
            /* iterate on the operands */
            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const Instr::operand_t *op = instr->getOperand(i);

                if (op->type == Instr::operandType::MREF && 
                    (instr->getMemOpType() == Instr::memOpType::GENERIC
                    || instr->getMemOpType() == Instr::memOpType::GLOBAL)) {
                    /* insert call to the instrumentation function with its
                     * arguments */
                    nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
                    /* predicate value */
                    nvbit_add_call_arg_pred_val(instr);
                    /* opcode id */
                    nvbit_add_call_arg_const_val32(instr, opcode_id);
                    /* memory reference 64 bit address */
                    nvbit_add_call_arg_mref_addr64(instr, mref_idx);
                    /* scope of operation */
                    nvbit_add_call_arg_const_val32(instr, getScope(instr));
                    /* load operation? */
                    nvbit_add_call_arg_const_val32(instr, (instr->isLoad() ? MASK_LOAD : 0) | 
                        (instr->isStore() ? MASK_STORE : 0) | (isStrong(instr) ? MASK_STRONG : 0) | 
                        (isCAS(instr) ? MASK_CAS : 0) | (isExch(instr) ? MASK_EXCH : 0));
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)metadata);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)&addrRangeStart);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)&mdArrayLen);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)counters);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)&channel_dev);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)parameters);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val32(instr, (uint32_t)instr->getSize());
                    mref_idx++;
                }
            }
        }
    }
    if(debug_out)
        instru_time += (double)std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start_time).count() / 1000.0;
}

static void update_rss()
{
    size_t free = 0, total = 0;
	CUDA_SAFECALL(hipMemGetInfo(&free, &total));
	if(maxGPUMem < total - free) {
	    maxGPUMem = total - free;
	    userGPUMem = dataSize;
	}
}

__global__ void flush_channel() {
    /* push memory access with negative cta id to communicate the kernel is
     * completed */
    mem_access_t ma;
    ma.warp_id = -1;
    channel_dev.push(&ma, sizeof(mem_access_t));

    /* flush channel */
    channel_dev.flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag || turned_off) return;

    if(cbid == API_CUDA_cuMemAlloc_v2 && is_exit) {
        cuMemAlloc_v2_params *p = (cuMemAlloc_v2_params *)params;
        dataSize += p->bytesize;
        printf("Allocated %lu bytes of memory at %llx, total mem %lu\n", p->bytesize, *p->dptr, dataSize);
        ptrSizes[*p->dptr] = p->bytesize;
    }
    
    else if(cbid == API_CUDA_cuMemAllocManaged && is_exit) {
        cuMemAllocManaged_params *p = (cuMemAllocManaged_params *)params;
        dataSize += p->bytesize;
        printf("Allocated %lu bytes of managed memory at %llx, total mem %lu\n", p->bytesize, *p->dptr, dataSize);
        ptrSizes[*p->dptr] = p->bytesize;        
    }
    
    else if((cbid == API_CUDA_cuMemAllocHost_v2) && is_exit) {
        cuMemAllocHost_v2_params *p = (cuMemAllocHost_v2_params *)params;
        dataSize += p->bytesize;
        printf("Allocated %lu bytes of host memory at %p, total mem %lu\n", p->bytesize, *p->pp, dataSize);
        ptrSizes[(hipDeviceptr_t)*p->pp] = p->bytesize;        
    }
        
    else if(cbid == API_CUDA_cuMemFree_v2 && is_exit) {
        cuMemFree_v2_params *p = (cuMemFree_v2_params *)params;
        size_t size = ptrSizes[p->dptr];
        dataSize -= size;
        ptrSizes.erase(p->dptr);
        printf("Freed %llx, total mem %lu\n", p->dptr, dataSize);        
    }
    
    else if(cbid == API_CUDA_cuMemFreeHost && is_exit) {
        cuMemFreeHost_params *p = (cuMemFreeHost_params *)params;
        size_t size = ptrSizes[(hipDeviceptr_t)p->p];
        dataSize -= size;
        ptrSizes.erase((hipDeviceptr_t)p->p);
        printf("Host freed %llx, total mem %lu\n", (hipDeviceptr_t)p->p, dataSize); 
        
    }

    else if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel ||
        cbid == API_CUDA_cuLaunchCooperativeKernel ||
        cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz) {
        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

        if (!is_exit) {
            instrument_function_if_needed(ctx, p->f);
            nvbit_enable_instrumented(ctx, p->f, true);
                int nregs;
                CUDA_SAFECALL(
                    hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

                int shmem_static_nbytes;
                CUDA_SAFECALL(
                    hipFuncGetAttribute(&shmem_static_nbytes,
                                       HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));
            printf(
                "Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
                "%d - shmem %d - cuda stream id %ld\n",
                nvbit_get_func_name(ctx, p->f), p->gridDimX, p->gridDimY,
                p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ, nregs,
                shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);
            if(debug_out) {
                start_time = std::chrono::high_resolution_clock::now();
            }
            skip_flag = true;
            if(started == true) {
                for(unsigned i = 0; i < TOTAL_CTRS; ++i)
                    if((check_locking || i != LOCKS) && (check_its || i != WARP_BAR))
                        CUDA_SAFECALL(hipFree(counters[i]));
                if(md_scale > 1) {
                    CUDA_SAFECALL(hipFree(metadata[WR_MD]));
                    CUDA_SAFECALL(hipFree(metadata[RD_MD]));
                }
            }
            
            started = true;
            
            if(md_scale > 1) {
                CUDA_SAFECALL(hipMallocManaged((void**)&metadata[WR_MD], sizeof(uint64_t) * roundUp(dataSize, granularity * md_scale)));
                CUDA_SAFECALL(hipMallocManaged((void**)&metadata[RD_MD], sizeof(uint64_t) * roundUp(dataSize, granularity * md_scale)));
                mdArrayLen = roundUp(dataSize, granularity * md_scale);
            }
            
            uint64_t NBLOCKS = p->gridDimX * p->gridDimY * p->gridDimZ;
            uint64_t NWARPS  = roundUp(p->blockDimX * p->blockDimY * p->blockDimZ, WARP_SIZE) * NBLOCKS;
            
            size_t free = 0, total = 0;
            CUDA_SAFECALL(hipMemGetInfo(&free, &total));
            
            if(managed || free < sizeof(BYTE) * (2 * NWARPS + NBLOCKS)) { 
                CUDA_SAFECALL(hipMallocManaged((void**)&counters[BARRIER],   sizeof(BYTE) * NBLOCKS));
                CUDA_SAFECALL(hipMallocManaged((void**)&counters[WARP_CTRS], sizeof(HWORD) * NWARPS * WARP_SIZE));
            }
            else {
                CUDA_SAFECALL(hipMalloc((void**)&counters[BARRIER],   sizeof(BYTE) * NBLOCKS));
                CUDA_SAFECALL(hipMalloc((void**)&counters[WARP_CTRS], sizeof(HWORD) * NWARPS * WARP_SIZE));
                free -= sizeof(BYTE) * (NBLOCKS) + sizeof(DWORD) * NWARPS;
            }            
            
            if(check_its) {
                if(managed || free < sizeof(BYTE) * NWARPS) {
                    CUDA_SAFECALL(hipMallocManaged((void**)&counters[WARP_BAR], sizeof(BYTE) * NWARPS));
                }
                else {
                    CUDA_SAFECALL(hipMalloc((void**)&counters[WARP_BAR], sizeof(BYTE) * NWARPS));
                    free -= sizeof(BYTE) * NWARPS;
                }                 
            }
            if(check_locking) {
                if(managed || free < sizeof(DWORD) * NWARPS * /*(lock_granularity == 0 ? 1 : */WARP_SIZE) {
                    CUDA_SAFECALL(hipMallocManaged((void**)&counters[LOCKS], sizeof(DWORD) * NWARPS * /*(lock_granularity == 0 ? 1 : */WARP_SIZE));
                }
                else {
                    CUDA_SAFECALL(hipMalloc((void**)&counters[LOCKS], sizeof(DWORD) * NWARPS * /*(lock_granularity == 0 ? 1 : */WARP_SIZE));
                    free -= sizeof(DWORD) * NWARPS * /*(lock_granularity == 0 ? 1 : */WARP_SIZE;
                }
            }
            
            CUDA_SAFECALL(hipMemset(counters[BARRIER],   0, sizeof(BYTE) * NBLOCKS));
            CUDA_SAFECALL(hipMemset(counters[WARP_CTRS], 0, sizeof(HWORD) * NWARPS * WARP_SIZE));
            if(check_its)
                CUDA_SAFECALL(hipMemset(counters[WARP_BAR], 0, sizeof(BYTE) * NWARPS));
            if(check_locking)
                CUDA_SAFECALL(hipMemset(counters[LOCKS], 0, sizeof(DWORD) * NWARPS * WARP_SIZE));
            uint64_t now_used = 0;
            CUDA_SAFECALL(hipMemGetInfo(&free, &total));
            free += prev_used;
            for(auto i = ptrSizes.begin(); i != ptrSizes.end(); ++i) {
                uint64_t offset = (i->first / granularity) % mdArrayLen;
                if(free > 2 * sizeof(uint64_t) * roundUp(i->second, granularity)) {
                    // Set md to 0. If data wraps around array, split into two memsets
                    if(roundUp(i->second, granularity) + offset < mdArrayLen) {
                        CUDA_SAFECALL(hipMemset((uint64_t*)metadata[WR_MD] + offset, 0, sizeof(uint64_t) * roundUp(i->second, granularity)));
                        CUDA_SAFECALL(hipMemset((uint64_t*)metadata[RD_MD] + offset, 0, sizeof(uint64_t) * roundUp(i->second, granularity)));
                        now_used += 2 * sizeof(uint64_t) * roundUp(i->second, granularity);
                        if(free > 2 * sizeof(uint64_t) * roundUp(i->second, granularity))
                            free -= 2 * sizeof(uint64_t) * roundUp(i->second, granularity);
                        else
                            free = 0;
                    }
                    else {
                        // Data size exceeds array size. Just set everything to zero and leave
                        if(roundUp(i->second, granularity) >= mdArrayLen) {
                            CUDA_SAFECALL(hipMemset((uint64_t*)metadata[WR_MD], 0, sizeof(uint64_t) * mdArrayLen));
                            CUDA_SAFECALL(hipMemset((uint64_t*)metadata[RD_MD], 0, sizeof(uint64_t) * mdArrayLen));
                            now_used = 2 * mdArrayLen;
                            if(free > 2 * mdArrayLen)
                                free -= 2 * mdArrayLen;
                            else
                                free = 0;
                            break;
                        }
                        
                        uint64_t extra = roundUp(i->second, granularity) - (mdArrayLen - offset);
                        CUDA_SAFECALL(hipMemset((uint64_t*)metadata[WR_MD] + offset, 0, sizeof(uint64_t) * (mdArrayLen - offset)));
                        CUDA_SAFECALL(hipMemset((uint64_t*)metadata[RD_MD] + offset, 0, sizeof(uint64_t) * (mdArrayLen - offset)));
                        CUDA_SAFECALL(hipMemset((uint64_t*)metadata[WR_MD], 0, sizeof(uint64_t) * extra));
                        CUDA_SAFECALL(hipMemset((uint64_t*)metadata[RD_MD], 0, sizeof(uint64_t) * extra));
                        now_used += 2 * sizeof(uint64_t) * roundUp(i->second, granularity);
                        if(free > 2 * sizeof(uint64_t) * roundUp(i->second, granularity))
                            free -= 2 * sizeof(uint64_t) * roundUp(i->second, granularity);
                        else
                            free = 0;
                    }
                }
                else {
                    printf("Data too large, memset on CPU. Free %ld, total %ld, needed %ld, used %ld\n", free, total, 2 * sizeof(uint64_t) * roundUp(i->second, granularity), prev_used);
                    // Set md to 0. If data wraps around array, split into two memsets
                    if(roundUp(i->second, granularity) + offset < mdArrayLen) {
                        memset((uint64_t*)metadata[WR_MD] + offset, 0, sizeof(uint64_t) * roundUp(i->second, granularity));
                        memset((uint64_t*)metadata[RD_MD] + offset, 0, sizeof(uint64_t) * roundUp(i->second, granularity));
                    }
                    else {
                        // Data size exceeds array size. Just set everything to zero and leave
                        if(roundUp(i->second, granularity) >= mdArrayLen) {
                            memset((uint64_t*)metadata[WR_MD], 0, sizeof(uint64_t) * mdArrayLen);
                            memset((uint64_t*)metadata[RD_MD], 0, sizeof(uint64_t) * mdArrayLen);
                            break;
                        }
                        
                        uint64_t extra = roundUp(i->second, granularity) - (mdArrayLen - offset);
                        memset((uint64_t*)metadata[WR_MD] + offset, 0, sizeof(uint64_t) * (mdArrayLen - offset));
                        memset((uint64_t*)metadata[RD_MD] + offset, 0, sizeof(uint64_t) * (mdArrayLen - offset));
                        memset((uint64_t*)metadata[WR_MD], 0, sizeof(uint64_t) * extra);
                        memset((uint64_t*)metadata[RD_MD], 0, sizeof(uint64_t) * extra);
                    }
                }
            }
            prev_used = now_used;
            skip_flag = false;
            
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if(err != hipSuccess) {
                printf("CUDA error (%d): %s\n", err, hipGetErrorName (err));
                fflush(stdout);
                assert(false);
            }
            recv_thread_receiving = true;
            if(debug_out) {
                setup_time += (double)std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start_time).count() / 1000.0;
                start_kernel = std::chrono::high_resolution_clock::now();
            }
        } else {
            
            if(debug_out)           
                update_rss();
            /* make sure current kernel is completed */
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if(err != hipSuccess) {
                printf("CUDA error (%d): %s\n", err, hipGetErrorName (err));
                fflush(stdout);
                assert(false);
            }
            if(debug_out) {
                kernel_time += (double)std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start_kernel).count() / 1000.0;
                start_time = std::chrono::high_resolution_clock::now();
            }

            /* make sure we prevent re-entry on the nvbit_callback when issuing
             * the flush_channel kernel */
            skip_flag = true;

            /* issue flush of channel so we are sure all the memory accesses
             * have been pushed */
            flush_channel<<<1, 1>>>();
            hipDeviceSynchronize();
            err = hipGetLastError();
            if(err != hipSuccess) {
                printf("CUDA error (%d): %s\n", err, hipGetErrorName (err));
                fflush(stdout);
                assert(false);
            }

            /* unset the skip flag */
            skip_flag = false;

            /* wait here until the receiving thread has not finished with the
             * current kernel */
            while (recv_thread_receiving) {
                pthread_yield();
            }
            if(debug_out)
                thread_time += (double)std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start_time).count() / 1000.0;
        }
    }
}

void *recv_thread_fun(void *) {
    char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
    hipStream_t pStream;
    bool flushed = false;
    while (recv_thread_started) {
        //if(flushed) // Timed out and wrapped around
        //    assert(false);
        auto end = std::chrono::high_resolution_clock::now();
        if(!flushed && timeout > 0 && std::chrono::duration_cast<std::chrono::seconds>(end - start).count() > timeout) {
            printf("\nKernel timed out.\n");
            fflush(stdout);
            int highestPriority;
            hipDeviceGetStreamPriorityRange (NULL, &highestPriority );
            hipStreamCreateWithPriority ( &pStream, hipStreamNonBlocking, highestPriority );
            skip_flag = true;
            flush_channel<<<1, 1, 0, pStream>>>();
            flushed = true;
            skip_flag = false;
        }
        
        uint32_t num_recv_bytes = 0;
        if (recv_thread_receiving &&
            (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
                0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                mem_access_t *ma =
                    (mem_access_t *)&recv_buffer[num_processed_bytes];

                /* when we get this cta_id_x it means the kernel has completed */
                if (ma->warp_id == -1) {
                    recv_thread_receiving = false;
                    if(flushed) // Due to timeout
                        assert(false);
                    break;
                }
                num_processed_bytes += sizeof(mem_access_t);
                
                if(id_to_opcode_map.find(ma->opcode_id) == id_to_opcode_map.end())
                    continue;
                
                printf("\n");
                switch(ma->reason) {
                    case RACE_BFENCE: printf("Race: Missing blkfence");    break;
                    case RACE_GFENCE: printf("Race: Missing gpufence");    break;
                    case RACE_STRONG: printf("Race: Missing strong op");   break;
                    case RACE_ATOMIC: printf("Race: Improper atom scope"); break;
                    case RACE_LOCK:   printf("Race: Missing lock");        break;
                    case RACE_ITS:    printf("Race: Missing warpsync");    break;
                }
                printf("\n");
                
                // Get line info and file details
                std::string p = id_to_opcode_map[ma->opcode_id];
                
                printf("%s - ", p.c_str());
                printf("(TID %lu, ", ma->warp_id);
                id_to_opcode_map.erase(ma->opcode_id);
                printf("%lx)", ma->addr);
                printf("\n");
                uint64_t md = ma->write_md;
                printf("Write: M(%lu), BShr(%lu), GShr(%lu), Atom(%lu), Scope(%lu), Str(%lu), TID(%lu), GF(%lu), BF(%lu), Bar(%lu), WBar(%lu), Locks(%lx)\n",
                    getBit(md, BIT1_MOD), getBit(md, BIT1_BSHR), getBit(md, BIT1_GSHR), getBit(md, BIT1_ATOMIC), getBit(md, BIT1_SCOPE), 
                    getBit(md, BIT_STRONG), getBits(md, BIT_TID, SZ_TID), getBits(md, BIT_GFENCE, SZ_GFENCE), getBits(md, BIT_BFENCE, SZ_BFENCE), 
                    getBits(md, BIT_BAR, SZ_BAR), getBits(md, BIT_WBAR, SZ_WBAR), getBits(md, BIT_LOCKS, SZ_LOCKS));
                
                md = ma->read_md;
                printf("Read: Tag(%lu), Str(%lu), TID(%lu), GF(%lu), BF(%lu), Bar(%lu), WBar(%lu), Locks(%lx)\n",
                    getBits(md, BIT2_TAG, SZ_TAG), getBit(md, BIT_STRONG), getBits(md, BIT_TID, SZ_TID), getBits(md, BIT_GFENCE, SZ_GFENCE), 
                    getBits(md, BIT_BFENCE, SZ_BFENCE), getBits(md, BIT_BAR, SZ_BAR), getBits(md, BIT_WBAR, SZ_WBAR), getBits(md, BIT_LOCKS, SZ_LOCKS)); 
                
                printf("OGF(%lu), OBF(%lu), OBar(%lu), Heldlock(%lu)\n", getBits(ma->extra, 0, 8), getBits(ma->extra, 8, 8), getBits(ma->extra, 16, 8), getBits(ma->extra, 24, 16));
                
                fflush(stdout);
                if(race_exit)
                    assert(false);
            }
        }
    }
    free(recv_buffer);
    return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    if(!turned_off && !recv_thread_started) {
        recv_thread_started = true;
        channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
        pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
    }
    
    start = std::chrono::high_resolution_clock::now();
    skip_flag = true;
    hipMemcpy(&parameters[BYTE_GRAN], &granularity, sizeof(uint32_t), hipMemcpyHostToDevice);
    uint32_t val = ((check_locking ? MASK_CHECK_LOCKS : 0) | /*(lock_granularity ? MASK_LOCK_GRAN : 0) |*/ (check_its ? MASK_CHECK_ITS : 0));
    hipMemcpy(&parameters[OPTIONS], &val, sizeof(uint32_t), hipMemcpyHostToDevice);
    
    size_t free = 0, total = 0;
    CUDA_SAFECALL(hipMemGetInfo(&free, &total));
    if(md_scale == 1) {
        if(managed) {
            CUDA_SAFECALL(hipMallocManaged((void**)&metadata[WR_MD], sizeof(uint64_t) * roundUp(total, granularity)));
            CUDA_SAFECALL(hipMallocManaged((void**)&metadata[RD_MD], sizeof(uint64_t) * roundUp(total, granularity)));
            mdArrayLen = roundUp(total, granularity);
        }
        else { // Redundant condition, to be removed in future.
            CUDA_SAFECALL(hipMalloc((void**)&metadata[WR_MD], sizeof(uint64_t) * roundUp(total / 8, granularity)));
            CUDA_SAFECALL(hipMalloc((void**)&metadata[RD_MD], sizeof(uint64_t) * roundUp(total / 8, granularity)));
            mdArrayLen = roundUp(total / 8, granularity);
        }
        hipDeviceSynchronize();
    }
    
    skip_flag = false;
    if(debug_out) {
        init_time += (double)std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() / 1000.0;
    }
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (recv_thread_started) {
        recv_thread_started = false;
    }
    char linkname[1024] = "<unknown>";
    int len = readlink("/proc/self/exe", linkname, sizeof(linkname));
    const char* name = strrchr(linkname, '/');
    if(name == NULL) name = linkname; else name += 1;
    fflush(stdout);
    fflush(stderr);
    
    skip_flag = true;
    if(started == true) {
        for(unsigned i = 0; i < TOTAL_CTRS; ++i)
            if((check_locking || i != LOCKS) && (check_its || i != WARP_BAR))
                CUDA_SAFECALL(hipFree(counters[i]));
        CUDA_SAFECALL(hipFree(metadata[WR_MD]));
        CUDA_SAFECALL(hipFree(metadata[RD_MD]));
    }
    skip_flag = false;
    if(recv_thread_started) {
        pthread_join(recv_thread, NULL);
    }
    auto end = std::chrono::high_resolution_clock::now();
    if(debug_out) {
        printf("TIME MS %s %lf %s\n", name, (double)std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000.0, 
            (turned_off ? "DISABLED" : "ENABLED"));
        printf("MAXGPUMEM\t%s\t%lu\n", name, maxGPUMem);
        printf("USERGPUMEM\t%s\t%lu\n", name, userGPUMem);
        printf("\tInit\tInstrument\tThread\tSetup\tKernel\nBREAKDOWN\t%f\t%f\t%f\t%f\t%f\n", init_time, instru_time, thread_time, setup_time, kernel_time);
    }
}
