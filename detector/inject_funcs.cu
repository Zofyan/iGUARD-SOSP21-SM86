#include "hip/hip_runtime.h"
/********************************************************************************************
 * Copyright (c) 2021 Indian Institute of Science
 * All rights reserved.
 *
 * Developed by:    Aditya K Kamath
 *                  Computer Systems Lab
 *                  Indian Institute of Science
 *                  https://csl.csa.iisc.ac.in/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of Computer Systems Lab, Indian Institute of Science, 
 *        nor the names of its contributors may be used to endorse or promote products 
 *        derived from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 ********************************************************************************************/

#include <stdint.h>
#include <stdio.h>

// Comment to remove printfs
//#define DEBUG

#include "utils/utils.h"
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"

__device__ int global_lock = 0;

extern "C" __device__ __noinline__ void instrument_fence(int pred, scope_t scope, uint64_t fenceId,
    uint64_t locks, uint64_t parameters) 
{
    if (!pred) {
        return;
    }
    
    unsigned mask = __activemask();
    
    /* Get actual array */
    fenceId = *(uint64_t*)fenceId;
    locks = *(uint64_t*)locks;
    
    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    // Global threadId i.e. across all blocks
    uint64_t g_tid = tid + bid * blockDim.x * blockDim.y * blockDim.z;
    
    tid %= WARP_SIZE;
    // (mask - 1) & mask -> Unset last bit
    // ^ mask -> Unset all bits except last bit
    unsigned selectedThread = ((mask - 1) & mask) ^ mask;
    
    // Activate locks if needed
    if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_LOCKS)) {
        uint64_t lock_table = atomicAdd(&((ULL*)locks)[g_wid], 0);
        bool moved = false;
        if(getBit(lock_table, LKBIT_MOVED)) {// Use thread level 
            lock_table = atomicAdd(&((ULL*)locks)[g_tid], 0);
            moved = true;
        }
        if(getBits(lock_table, 0, LKBIT_MOVED) != 0) { // Skip in common case
            for(uint32_t i = 0; i < LKS_PER_THD; ++i) {
                if(scope != SCOPE_CTA || getBit(lock_table, LOCK_SIZE * i + LKBIT_SCOPE))
                    setBit(lock_table, LOCK_SIZE * i + LKBIT_ACTIVE); // Mark locks active
            }
            
            // Write back
            if(moved)
                atomicExch(&((ULL*)locks)[g_tid], lock_table);
            else if((1 << tid) & selectedThread)
                atomicExch(&((ULL*)locks)[g_wid], lock_table);
        }
    }
    
    g_tid = (g_wid << 5) | (tid & ((ONE << 5) - ONE));
    // Only last thread updates
    //if((1 << tid) & selectedThread) {
        switch(scope) {
            case SCOPE_NONE:
            case SCOPE_GPU:
            case SCOPE_SYS:
                ++(((BYTE*)fenceId)[sizeof(HWORD) * g_tid + GPU_FENCE]);
            break;
            case SCOPE_CTA:
                ++(((BYTE*)fenceId)[sizeof(HWORD) * g_tid + BLK_FENCE]);
            break;
        }
	    debug_printf("WID %lu: %s scope fence; blk: %d, dev: %d\n", g_tid, scopeToStr(scope), 
	        ((BYTE*)fenceId)[sizeof(HWORD) * g_tid + BLK_FENCE], ((BYTE*)FenceId)[sizeof(HWORD) * g_tid + GPU_FENCE]);
    //}
    // Force warp to wait until update complete
    __syncwarp(mask);
}

extern "C" __device__ __noinline__ void instrument_barrier(int pred, uint64_t barrierId) 
{
    if (!pred) {
        return;
    }
    
    /* Get actual array */
    barrierId = *(uint64_t*)barrierId;    
    
    int tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // Have only single warp update barrierID
    if(tid < WARP_SIZE) {
        int bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
        unsigned mask = __activemask();
        // (mask - 1) & mask -> Unset last bit
        // ^ mask -> Unset all bits except last bit
        unsigned selectedThread = ((mask - 1) & mask) ^ mask;
        // Only last thread updates
        if((1 << tid) & selectedThread) {
            ++(((BYTE*)barrierId)[bid]);
	        debug_printf("BID %d: Barrier; counter %d\n", bid, ((BYTE*)barrierId)[bid]);
        }
    }
    // Have other threads wait until update is received
    __syncthreads();
}

extern "C" __device__ __noinline__ void instrument_warp_bar(int pred, uint64_t warpBarrierId) 
{
    if (!pred) {
        return;
    }
    unsigned mask = __activemask();
    warpBarrierId = *(uint64_t*)warpBarrierId;
    
    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    
    tid %= WARP_SIZE;    
    // (mask - 1) & mask -> Unset last bit
    // ^ mask -> Unset all bits except last bit
    unsigned selectedThread = ((mask - 1) & mask) ^ mask;
    // Only last thread updates
    if((1 << tid) & selectedThread) {
        ++(((BYTE*)warpBarrierId)[g_wid]);
	    debug_printf("WID %lu: Warp barrier; counter %d\n", g_wid,
	        ((BYTE*)warpBarrierId)[g_wid]);
    }
    __syncwarp(mask);
}

__device__ __inline__ void print_instr(uint32_t op_mask, scope_t scope, uint64_t addr, uint64_t offset)
{
    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    tid %= WARP_SIZE;
    unsigned mask = __activemask();
    // (mask - 1) & mask -> Unset last bit
    // ^ mask -> Unset all bits except last bit
    unsigned selectedThread = ((mask - 1) & mask) ^ mask;
    // Only last thread updates
    if((1 << tid) & selectedThread) {
        if(hasMask(op_mask, MASK_ATOMIC)) {
            debug_printf("WID %lu: %s scope atomic at %lx, offset %lx\n", g_wid, scopeToStr(scope), addr, offset);
        }
        
        else if(hasMask(op_mask, MASK_LOAD)) {
            debug_printf("WID %lu: %s %s scope load at %lx, offset %lx\n", g_wid, 
                (hasMask(op_mask, MASK_STRONG) ? "Strong" : "Weak"), scopeToStr(scope), addr, offset);
        }
        
        else if(hasMask(op_mask, MASK_STORE)) {
            debug_printf("WID %lu: %s %s scope store at %lx, offset %lx\n", g_wid, 
                (hasMask(op_mask, MASK_STRONG) ? "Strong" : "Weak"), scopeToStr(scope), addr, offset);
        }
    }
}

__device__ __inline__ void print_md(uint64_t md, uint64_t read_md, uint64_t g_wid, uint64_t filter, uint64_t offset, BYTE GF, BYTE BF, BYTE OWB)
{
    unsigned tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    tid %= WARP_SIZE;
    unsigned mask = __activemask();
    // (mask - 1) & mask -> Unset last bit
    // ^ mask -> Unset all bits except last bit
    unsigned selectedThread = __ffs(mask) - 1;
    // Only last thread updates
    if(tid == selectedThread)
        debug_printf("WID %lu (%x): (%lx) V(%lu), M(%lu), GS(%lu), BS(%lu), Atom(%lu), Scope(%lu), Str(%lu), TID_W(%lu), TID_R(%lu), GF(%lu), BF(%lu), Bar(%lu), WBar(%lu), Locks(%lx); Locks held(%lx), OWGF(%d), OWBF(%d), OBAR(%d)\n", 
            g_wid, mask, offset, getBit(md, BIT1_VALID), getBit(md, BIT1_MOD), getBit(md, BIT1_GSHR), getBit(md, BIT1_BSHR), getBit(md, BIT1_ATOMIC), getBit(md, BIT1_SCOPE), 
            getBit(md, BIT_STRONG), getBits(md, BIT_TID, SZ_TID), getBits(read_md, BIT_TID, SZ_TID), getBits(md, BIT_GFENCE, SZ_GFENCE), getBits(md, BIT_BFENCE, SZ_BFENCE), 
            getBits(md, BIT_BAR, SZ_BAR), getBits(md, BIT_WBAR, SZ_WBAR), (getBits(md, BIT_LOCKS, SZ_LOCKS) << 8) | getBits(read_md, BIT_LOCKS, SZ_LOCKS), filter, GF, BF, OWB); 
}

__device__ __inline__ void setup_lock(void *locks, uint32_t op_mask, uint64_t addr, scope_t scope, uint64_t parameters, uint32_t threadMask)
{
    // Do we need to check locks?
    if(!hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_LOCKS)) return;
    // Check if atomic
    if(!hasMask(op_mask, MASK_ATOMIC)) return;
    // Check if CAS or Exch
    if(!hasMask(op_mask, MASK_CAS) && !hasMask(op_mask, MASK_EXCH)) return;
    
    unsigned mask = __activemask();

    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    // Global threadId i.e. across all blocks
    uint64_t g_tid = tid + bid * blockDim.x * blockDim.y * blockDim.z;
    
    if(hasMask(op_mask, MASK_CAS)) {
    
        uint64_t lock_table = atomicAdd(&((ULL*)locks)[g_wid], 0);
    
        bool moved = false;

        // More than one thread performing op
        if((threadMask - 1) & threadMask) {
            lock_table = atomicAdd(&((ULL*)locks)[g_tid], 0);
            moved = true;
            // Dynamically switch from warp level to thread level
            if(!getBit(lock_table, LKBIT_MOVED)) {
                lock_table = 0; // Reset table for first use
                setBit(lock_table, LKBIT_MOVED);
                // Set moved for warp-level table
                atomicOr(&((ULL*)locks)[g_wid], (ONE << (uint64_t)LKBIT_MOVED));
            }
        }

        else if(getBit(lock_table, LKBIT_MOVED)) { // Use thread level
            lock_table = atomicAdd(&((ULL*)locks)[g_tid], 0);
            moved = true;
        }

        // Get lower order bits of address. Atomics use
        // 4/8-byte aligned variables so ignore lower 2 bits
        uint64_t addr_bits = getBits(addr, 2, LKDATA_SIZE);
        // See if lock already exists in table
        bool success = false;
        for(uint32_t i = 0; i < LKS_PER_THD; ++i) {
            if(getBits(lock_table, LOCK_SIZE * i, LKDATA_SIZE) == addr_bits) { // Same lock
                setBit(lock_table,  LOCK_SIZE * i + LKBIT_ACTIVE, 0); // Mark inactive
                setBit(lock_table,  LOCK_SIZE * i + LKBIT_VALID, 1); // Mark valid
                setBit(lock_table,  LOCK_SIZE * i + LKBIT_SCOPE, scope == SCOPE_CTA); // Set scope
                success = true;
                break;
            }
        }
        // Find inactive slot and insert
        if(!success) {
            for(uint32_t i = 0; i < LKS_PER_THD; ++i) {
                if(!getBit(lock_table,  LOCK_SIZE * i + LKBIT_VALID)) { // Not valid
                    setBits(lock_table, LOCK_SIZE * i,  LKDATA_SIZE, addr_bits);
                    setBit(lock_table,  LOCK_SIZE * i + LKBIT_ACTIVE, 0); // Mark inactive
                    setBit(lock_table,  LOCK_SIZE * i + LKBIT_VALID, 1); // Mark valid
                    setBit(lock_table,  LOCK_SIZE * i + LKBIT_SCOPE, scope == SCOPE_CTA); // Set scope
                    success = true;
                    break;
                }
            }
        }
        if(!success) {
            // Replace arbitrary lock
            uint32_t i = addr_bits % LKS_PER_THD;
            setBits(lock_table, LOCK_SIZE * i,  LKDATA_SIZE, addr_bits);
            setBit(lock_table,  LOCK_SIZE * i + LKBIT_ACTIVE, 0); // Mark inactive
            setBit(lock_table,  LOCK_SIZE * i + LKBIT_VALID, 1); // Mark valid
            setBit(lock_table,  LOCK_SIZE * i + LKBIT_SCOPE, scope == SCOPE_CTA); // Set scope
            success = true;
        }
        
        tid %= WARP_SIZE;
        // (mask - 1) & mask -> Unset last bit
        // ^ mask -> Unset all bits except last bit
        unsigned selectedThread = ((mask - 1) & mask) ^ mask;
        // Write back
        if(moved)
            atomicExch(&((ULL*)locks)[g_tid], lock_table);
        else if((1 << tid) & selectedThread)
            atomicExch(&((ULL*)locks)[g_wid], lock_table);
    }
    else if(hasMask(op_mask, MASK_EXCH)) {
        //debug_printf("WID %lu: EXCH on %lx\n", g_wid, addr);
        uint64_t lock_table = atomicAdd(&((ULL*)locks)[g_wid], 0);
    
        bool moved = false;
        if(getBit(lock_table, LKBIT_MOVED)) {// Use thread level 
            lock_table = atomicAdd(&((ULL*)locks)[g_tid], 0);
            moved = true;
        }

        // Get lower order bits of address. Atomics use
        // 4/8-byte aligned variables so ignore lower 2 bits
        uint64_t addr_bits = getBits(addr, 2, LKDATA_SIZE);
        for(uint32_t i = 0; i < LKS_PER_THD; ++i) {
            if(getBits(lock_table, LOCK_SIZE * i, LKDATA_SIZE) == addr_bits && 
                (scope != SCOPE_CTA || getBit(lock_table, LOCK_SIZE * i + LKBIT_SCOPE)))
                setBit(lock_table, LOCK_SIZE * i + LKBIT_VALID, 0); // Mark matching locks invalid
        }
        
        tid %= WARP_SIZE;
        // (mask - 1) & mask -> Unset last bit
        // ^ mask -> Unset all bits except last bit
        unsigned selectedThread = ((mask - 1) & mask) ^ mask;
        // Write back
        if(moved)
            atomicExch(&((ULL*)locks)[g_tid], lock_table);
        else if((1 << tid) & selectedThread)
            atomicExch(&((ULL*)locks)[g_wid], lock_table);
    }
}

__device__ __inline__ uint64_t get_bloom_filter(void *locks, uint64_t parameters)
{
    if(!hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_LOCKS)) return 0;
    
    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    // Global threadId i.e. across all blocks
    uint64_t g_tid = tid + bid * blockDim.x * blockDim.y * blockDim.z;
    
    uint64_t lock_table = atomicAdd(&((ULL*)locks)[g_wid], 0);
    if(getBit(lock_table, LKBIT_MOVED)) {// Use thread level 
        lock_table = atomicAdd(&((ULL*)locks)[g_tid], 0);
    }
    
    if(getBits(lock_table, 0, LKBIT_MOVED) == 0) return 0;
    
    uint64_t filter = 0;
    for(int i = 0; i < LKS_PER_THD; ++i) {
        // If valid and active
        if(getBit(lock_table, i * LOCK_SIZE + LKBIT_VALID) && 
           getBit(lock_table, i * LOCK_SIZE + LKBIT_ACTIVE)) {
            filter |= (ONE << getBits(lock_table, i * LOCK_SIZE, 3));
            filter |= (ONE << ((uint64_t)3 + getBits(lock_table, i * LOCK_SIZE + 3, 3)));
        }
    }
    return filter;
}

__device__ __inline__ void set_bloom_filter(uint64_t &write_md, uint64_t &read_md, void *locks, uint64_t parameters)
{
    uint64_t filter = get_bloom_filter(locks, parameters);
    filter &= (getBits(write_md, BIT_LOCKS, SZ_LOCKS) << 8) | getBits(read_md, BIT_LOCKS, SZ_LOCKS);
    setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
    setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
}

__device__ __inline__ void setup_metadata(uint64_t &write_md, uint64_t &read_md, void **counters, uint32_t op_mask, scope_t scope, uint64_t parameters)
{
    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    // Global threadId, aligned to warp size
    uint64_t g_tid = (g_wid << 5) | (tid & ((ONE << 5) - ONE));
    
    BYTE barrier   = ((BYTE*)(counters[BARRIER]))  [bid];
    HWORD warp_ctrs = ((HWORD*)(counters[WARP_CTRS]))[g_tid];
    BYTE gpu_fence = getBits(warp_ctrs, GPU_FENCE * SZ_CTR, SZ_CTR);
    BYTE blk_fence = getBits(warp_ctrs, BLK_FENCE * SZ_CTR, SZ_CTR);
    BYTE warp_bar;
    if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_ITS))
        warp_bar  = ((BYTE*)(counters[WARP_BAR]))[g_wid];
    
    setBit (write_md, BIT1_ATOMIC, hasMask(op_mask, MASK_ATOMIC));
    setBit (write_md, BIT1_SCOPE,  scope == SCOPE_CTA);
    // On store set last writer
    if(hasMask(op_mask, MASK_STORE)) {
        setBit (write_md, BIT1_MOD,    1);
        //setBit (write_md, BIT_STRONG,  hasMask(op_mask, MASK_STRONG));
        setBits(write_md, BIT_TID,     SZ_TID,    g_tid);
        setBits(write_md, BIT_GFENCE,  SZ_GFENCE, gpu_fence);
        setBits(write_md, BIT_BFENCE,  SZ_BFENCE, blk_fence);
        setBits(write_md, BIT_BAR,     SZ_BAR,    barrier);
        if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_ITS)) {
            setBits(write_md, BIT_WBAR, SZ_WBAR, warp_bar);
        }
    }
    // Set last accessor
    //setBit (read_md, BIT_STRONG, hasMask(op_mask, MASK_STRONG));
    setBits(read_md, BIT_TID,    SZ_TID,    g_tid);
    setBits(read_md, BIT_GFENCE, SZ_GFENCE, gpu_fence);
    setBits(read_md, BIT_BFENCE, SZ_BFENCE, blk_fence);
    setBits(read_md, BIT_BAR,    SZ_BAR,    barrier);
    if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_ITS)) {
        setBits(read_md, BIT_WBAR, SZ_WBAR, warp_bar);
    }
}

/************************************************************************************************************
 *  CONDITIONS FOR SAFE ACCESS
 *  (a) First access   - md.Modified && md.BlkShared && md.DevShared
 *  (b) Program order  - md.WarpID == WarpID && md.BlockID == BlockID && !md.BlkShared && !md.DevShared
 *  (c) Barrier        - BlockID == md.BlockID && BarrierID != md.BarrierID && !md.DevShared
 *  
 *  CONDITIONS FOR RACEY ACCESS
 *  (a) Missing blkfence  - md.Modified && md.BlockID == BlockID && md.BlkFenceID == fFile.BlkFenceID 
 *                          && md.DevFenceID == fFile.DevFenceID
 *  (b) Missing devfence  - md.Modified && md.BlockID != BlockID && md.DevFenceID == fFile.DevFenceID
 *  (c) Not strong access - !md.Strong OR !Strong
 *  (d) Scoped atomic     - md.IsAtom && md.Scope == BLOCK && md.BlockID != BlockID
 *  (e) Missing lock      - intersect_locks().empty()
 ***********************************************************************************************************/

__device__ __inline__ uint32_t do_racecheck(uint64_t &write_md, uint64_t &read_md, uint64_t offset, void **counters, 
    uint32_t op_mask, scope_t scope, uint64_t &extra, uint64_t parameters, unsigned threadMask)
{
    uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
    // Local threadId, i.e. within a single block
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
    // Local warpId, i.e. within a single block
    uint64_t wid = tid / WARP_SIZE;
    // BlockId
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
    // Global warpId i.e. across all blocks
    uint64_t g_wid = wid + bid * WARPS_PER_BLK;
    // Global threadId, aligned to warp size
    uint64_t g_tid = (g_wid << 5) | (tid & ((ONE << 5) - ONE));
    
    BYTE barrier  = ((BYTE*) (counters[BARRIER]))  [bid];
    void *locks   = ((void**)(counters[LOCKS]));
    uint64_t filter = get_bloom_filter(locks, parameters);
    
    /* CONDITIONS FOR SAFE ACCESS */
    uint64_t *md;
    // Check if races with last access on write
    if(hasMask(op_mask, MASK_STORE))
        md = &read_md;
    // Check if races with last write on read
    else
        md = &write_md;

    bool SAME_BLK = (getBits(*md, BIT_WID, SZ_WID) / WARPS_PER_BLK == getBits(g_wid, 0, SZ_WID) / WARPS_PER_BLK);
    bool SAME_WRITE_BLK = (getBits(write_md, BIT_WID, SZ_WID) / WARPS_PER_BLK == getBits(g_wid, 0, SZ_WID) / WARPS_PER_BLK);
    bool SAME_READ_BLK = (getBits(read_md, BIT_WID, SZ_WID) / WARPS_PER_BLK == getBits(g_wid, 0, SZ_WID) / WARPS_PER_BLK);
    
    uint64_t other_warp = getBits(*md, BIT_WID, SZ_WID);
    uint64_t other_thread = getBits(*md, BIT_TID, SZ_TID);
    // Not first access. Get counters of other accessor
    HWORD warp_ctrs = ((HWORD*)(counters[WARP_CTRS]))[other_thread];
    uint64_t gpu_fence = getBits(warp_ctrs, GPU_FENCE * SZ_CTR, SZ_CTR);
    uint64_t blk_fence = getBits(warp_ctrs, BLK_FENCE * SZ_CTR, SZ_CTR);
    BYTE warp_bar;
    if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_ITS))
        warp_bar  = ((BYTE*)(counters[WARP_BAR]))[other_warp];
    setBits(extra, 0, 8, getBits(gpu_fence, 0, SZ_GFENCE));
    setBits(extra, 8, 8, getBits(blk_fence, 0, SZ_GFENCE));
    setBits(extra, 2 * 8, 8, getBits(barrier, 0, SZ_GFENCE));
    setBits(extra, 3 * 8, 16, filter);
    print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence, warp_bar);
    
    // First access, safe
    if(!getBit(write_md, BIT1_VALID)) {
        //debug_printf("WID %lu: First access to %lx, safe\n", g_wid, offset);
        write_md = 0;
        read_md = 0;
        setBit(write_md, BIT1_VALID);
        setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8)); // Setup locks
        setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter); // Setup locks
        return NO_RACE;
    }
    
    // Program order, safe
    if(getBits(*md, BIT_WID, SZ_WID) == getBits(g_wid, 0, SZ_WID) 
        && (!hasMask(op_mask, MASK_STORE) || (!getBit(write_md, BIT1_GSHR) && !getBit(write_md, BIT1_BSHR)))) {
        // Check for ITS
        if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_ITS)) {
            // Same thread, or currently in sync with thread, or synchronized at some point
            if(getBits(*md, BIT_TID, SZ_TID) == getBits(g_tid, 0, SZ_TID) || 
                ((ONE << getBits(*md, BIT_TID, 5)) & threadMask) || 
                (getBits(warp_bar, 0, SZ_WBAR) != getBits(*md, BIT_WBAR, SZ_WBAR)))
            {    
                if(SAME_BLK && getBits(*md, BIT_BAR, SZ_BAR) == barrier)           
                    set_bloom_filter(write_md, read_md, ((void**)counters)[LOCKS], parameters);
                else {
                    setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
                    setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
                }
                return NO_RACE;
            }
        }
        else {    
            if(SAME_BLK && getBits(*md, BIT_BAR, SZ_BAR) == getBits(barrier, BIT_BAR, SZ_BAR))           
                set_bloom_filter(write_md, read_md, ((void**)counters)[LOCKS], parameters);
            else {
                setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
                setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            }
            return NO_RACE;
        }            
    }
    // Barrier, safe
    if(SAME_BLK && getBits(*md, BIT_BAR, SZ_BAR) != barrier && !getBit(write_md, BIT1_GSHR)) {
        //debug_printf("WID %lu: Barrier to %lx, safe\n", g_wid, offset);
        setBit(write_md,  BIT1_BSHR, 0);
        setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
        setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
        return NO_RACE;
    }
    
    // Appropriately scoped atomic, safe
    if(getBit(write_md, BIT1_ATOMIC) && hasMask(op_mask, MASK_ATOMIC) && 
        (!getBit(write_md, BIT1_SCOPE) || SAME_WRITE_BLK)) {
        //debug_printf("WID %lu: %s scoped atomic to %lx, safe\n", g_wid, scopeToStr(scope), offset);
        set_bloom_filter(write_md, read_md, ((void**)counters)[LOCKS], parameters);  
        return NO_RACE;
    }
    
    bool MODIFIED = hasMask(op_mask, MASK_STORE) || getBit(write_md, BIT1_MOD);
    /* CONDITIONS FOR RACEY ACCESS */
    if(MODIFIED) {
        // Improperly scoped atomic, race
        if(getBit(write_md, BIT1_ATOMIC) && getBit(write_md, BIT1_SCOPE) && !SAME_WRITE_BLK) {
            //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
            debug_printf("WID %lu: improperly scoped atomic for %lx, race\n", g_wid, offset);
            // Set bits appropriately
            setBit(write_md, BIT1_MOD,  hasMask(op_mask, MASK_STORE));
            setBit(write_md, BIT1_GSHR, 0); // Reset bit for next detection
            setBit(write_md, BIT1_BSHR, 0); // Reset bit for next detection
            setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
            setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            return RACE_ATOMIC;
        }
        
        if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_ITS) && 
            getBits(*md, BIT_WID, SZ_WID) == getBits(g_wid, 0, SZ_WID) && 
            getBits(*md, BIT_GFENCE, SZ_GFENCE) == getBits(gpu_fence, 0, SZ_GFENCE) &&
            getBits(*md, BIT_BFENCE, SZ_BFENCE) == getBits(blk_fence, 0, SZ_BFENCE) && 
            !getBit(write_md, BIT1_BSHR) && !getBit(write_md, BIT1_GSHR)) {
            //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
            debug_printf("WID %lu: missing warpsync for %lx, race\n", g_wid, offset);
            // Set bits appropriately
            setBit(write_md, BIT1_MOD,  hasMask(op_mask, MASK_STORE));
            setBit(write_md, BIT1_GSHR, 0); // Reset bit for next detection
            setBit(write_md, BIT1_BSHR, 0); // Reset bit for next detection
            setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
            setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            return RACE_ITS;
        }
        
        // Missing blkFence, race
        if(SAME_BLK && getBits(*md, BIT_GFENCE, SZ_GFENCE) == getBits(gpu_fence, 0, SZ_GFENCE) &&
            getBits(*md, BIT_BFENCE, SZ_BFENCE) == getBits(blk_fence, 0, SZ_BFENCE)
            && !getBit(write_md, BIT1_GSHR)) {
            //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
            debug_printf("WID %lu: missing blkfence for %lx; OW=%lu, race\n", g_wid, offset, other_warp);
            // Set bits appropriately
            setBit(write_md, BIT1_MOD,  hasMask(op_mask, MASK_STORE));
            setBit(write_md, BIT1_GSHR, 0); // Reset bit for next detection
            setBit(write_md, BIT1_BSHR, 0); // Reset bit for next detection
            setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
            setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            return RACE_BFENCE;
        }
        // Missing gpufence, race
        else if(!SAME_BLK && getBits(*md, BIT_GFENCE, SZ_GFENCE) == getBits(gpu_fence, 0, SZ_GFENCE)) {
            //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
            debug_printf("WID %lu: missing gpufence for %lx; OW=%lu, race\n", g_wid, offset, other_warp);
            // Set bits appropriately
            setBit(write_md, BIT1_MOD,  hasMask(op_mask, MASK_STORE));
            setBit(write_md, BIT1_GSHR, 0); // Reset bit for next detection
            setBit(write_md, BIT1_BSHR, 0); // Reset bit for next detection
            setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
            setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            return RACE_GFENCE;    
        }
        // Missing strong ops, race
/*        else if(!getBit(*md, BIT_STRONG) || !hasMask(op_mask, MASK_STRONG)) {
            //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
            debug_printf("WID %lu: missing strong op for %lx, race\n", g_wid, offset);
            // Set bits appropriately
            setBit(write_md, BIT1_MOD,  hasMask(op_mask, MASK_STORE));
            setBit(write_md, BIT1_GSHR, 0); // Reset bit for next detection
            setBit(write_md, BIT1_BSHR, 0); // Reset bit for next detection
            setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
            setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            return RACE_STRONG;
        }*/
        
        // Missing locks
        uint64_t md_filter = (getBits(write_md, BIT_LOCKS, SZ_LOCKS) << 8) | getBits(read_md, BIT_LOCKS, SZ_LOCKS);
        if(hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CHECK_LOCKS) &&
            (md_filter != 0 || filter != 0) && (md_filter & filter) == 0) {
            
            //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
            debug_printf("WID %lu: missing lock for %lx, race\n", g_wid, offset);
            // Set bits appropriately
            setBit(write_md, BIT1_MOD,  hasMask(op_mask, MASK_STORE));
            setBit(write_md, BIT1_GSHR, 0); // Reset bit for next detection
            setBit(write_md, BIT1_BSHR, 0); // Reset bit for next detection
            setBits(write_md, BIT_LOCKS, SZ_LOCKS, (filter >> 8));
            setBits(read_md,  BIT_LOCKS, SZ_LOCKS, filter);
            return RACE_LOCK;        
        }        
    }
    
    uint64_t md_filter = (getBits(write_md, BIT_LOCKS, SZ_LOCKS) << 8) | getBits(read_md, BIT_LOCKS, SZ_LOCKS);
    if((md_filter != 0 || filter != 0) && (md_filter & filter) != 0) {
        //print_md(write_md, read_md, g_wid, filter, offset, gpu_fence, blk_fence);
        //debug_printf("WID %lu: properly locked for %lx\n", g_wid, offset);
        // Set bits appropriately
        set_bloom_filter(write_md, read_md, ((void**)counters)[LOCKS], parameters);  
        return NO_RACE;        
    }
    
    //debug_printf("WID %lu: No bad for %lx\n", g_wid, offset);
    // If modified, set not shared
    if(hasMask(op_mask, MASK_STORE)) {
        setBit(write_md, BIT1_GSHR, 0);
        setBit(write_md, BIT1_BSHR, 0);
    } else if (getBits(read_md, BIT_GFENCE, SZ_GFENCE) != getBits(gpu_fence, 0, SZ_GFENCE)) {
        setBit(write_md, BIT1_BSHR, 0);
        setBit(write_md, BIT1_GSHR, 0);
    } else if (SAME_READ_BLK && getBits(read_md, BIT_BFENCE, SZ_BFENCE) != getBits(blk_fence, 0, SZ_BFENCE)) {
        setBit(write_md, BIT1_BSHR, 0);
    } else if(getBits(write_md, BIT_WID, SZ_WID) / WARPS_PER_BLK != getBits(read_md, BIT_WID, SZ_WID) / WARPS_PER_BLK) {
        // Else if someone else has read set appropriate shared
        setBit(write_md, BIT1_GSHR, !SAME_READ_BLK || getBit(write_md, BIT1_GSHR)); // Set bit appropriately
        setBit(write_md, BIT1_BSHR, SAME_READ_BLK && !getBit(write_md, BIT1_GSHR)); // Reset bit for next detection
    }
    set_bloom_filter(write_md, read_md, ((void**)counters)[LOCKS], parameters);
    return NO_RACE;
}

extern "C" __device__ __noinline__ void instrument_mem(int pred, int opcode_id,
        uint64_t addr, scope_t scope, uint32_t op_mask, volatile uint64_t metadata,
        uint64_t addrStart, uint64_t mdArrayLen, uint64_t counters, uint64_t pchannel_dev, uint64_t parameters, int dataSize) 
{
    if (!pred)
        return;

    // BlockId
    unsigned mask = __activemask();
    
    // Perform contention optimizations?
    bool cont_opt = hasMask(((uint32_t*)parameters)[OPTIONS], MASK_CONTENT_OPT);
    
    uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);

    if(scope == SCOPE_CTA) {
        setup_lock(((void**)counters)[LOCKS], op_mask, addr ^ (bid << 2), scope, parameters, mask); // Acquire block-scoped lock
    }
    else {
        setup_lock(((void**)counters)[LOCKS], op_mask, addr, scope, parameters, mask); // Acquire global lock
        setup_lock(((void**)counters)[LOCKS], op_mask, addr ^ (bid << 2), scope, parameters, mask); // Acquire block-scoped lock
    }
    
    /* Get actual arrays */
    mdArrayLen  = *(uint64_t*)mdArrayLen;
    
    // Check if address belongs to global memory using PTX
    int is_global_mem;
    asm (".reg .pred p;\
        isspacep.global  p, %1;\
        selp.u32 %0,1,0,p;\
        ":"=r"(is_global_mem): "l"(addr));
        
    if(is_global_mem) {
        uint64_t internalOffset = 0;
		do{
		    uint64_t dataOffset = ((addr + internalOffset) / (uint64_t)((uint32_t*)parameters)[BYTE_GRAN]);
		    uint64_t mdOffset = dataOffset % mdArrayLen;
#ifdef DEBUG
		    print_instr(op_mask, scope, addr, mdOffset);
#endif
		    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
		    tid %= WARP_SIZE;
		    // If all threads are reading/atoming same location
		    // Have only one thread do race detection
		    unsigned mask2 = __activemask();
		    uint64_t oth_addr = __shfl_sync(mask2, addr, __ffs(mask2) - 1);
		    if(oth_addr != addr || !hasMask(op_mask, MASK_LOAD) || !cont_opt || (tid == __ffs(mask2) - 1)) {
		        uint32_t detected_race = NO_RACE;
		        DWORD race_read_md;
		        DWORD race_write_md;
		        DWORD extra = 0;
		        bool complete = false;
		        const unsigned WARPS = (blockDim.x * min(gridDim.x, 100)) / WARP_SIZE;
		        const unsigned BASE_DELAY = 100;
		        const unsigned MAX_DELAY = max(BASE_DELAY, WARPS * BASE_DELAY / 40);
		        unsigned delay = (cont_opt ? BASE_DELAY : 0);
		        do {
		            // -1 indicates unused, -2 indicates in-use
		            const DWORD reserved_data = (DWORD)-2;
		            DWORD read_md = atomicAdd(&((ULL**)metadata)[RD_MD][mdOffset], 0);
		            // Already reserved, can possibly do a delay here?
		            if(read_md == reserved_data) {
		                if(delay) {
		                    csleep(delay);
		                    delay *= 2;
		                    delay = min(delay, MAX_DELAY);
		                }
		                continue;
		            }
		            // Try to swap
		            if(atomicCAS(&((ULL**)metadata)[RD_MD][mdOffset], read_md, reserved_data) == read_md) {
		                __threadfence();
		                
		                DWORD write_md;
		                
		                uint64_t tag = getBits(dataOffset / mdArrayLen, 0, SZ_TAG);
		                // Tag mismatch, reset metadata
		                if(getBits(read_md, BIT2_TAG, SZ_TAG) != tag) {
		                    write_md = 0;
		                    read_md = 0;
		                }
		                else
		                    write_md = atomicAdd(&((ULL**)metadata)[WR_MD][mdOffset], 0);
		                
		                race_read_md = read_md;
		                race_write_md = write_md;
		                detected_race = do_racecheck(write_md, read_md, mdOffset, (void**)counters, op_mask, scope, extra, parameters, mask);
		                setup_metadata(write_md, read_md, (void**)counters, op_mask, scope, parameters);
		                setBits(read_md, BIT2_TAG, SZ_TAG, tag); // set tag
		                atomicExch(&((ULL**)metadata)[WR_MD][mdOffset], write_md);
		                __threadfence();
		                atomicExch(&((ULL**)metadata)[RD_MD][mdOffset], read_md);
		                complete = true;
		            }
		            else {
		                if(delay) {
		                    csleep(delay);
		                    delay *= 2;
		                    delay = min(delay, MAX_DELAY);
		                }
		            }
		        } while(!complete);
		        
		        if(detected_race != NO_RACE) {
		            unsigned mask = __activemask();
		            unsigned selectedThread = ((mask - 1) & mask) ^ mask;
		            // Only last thread updates
		            if((1 << tid) & selectedThread) {
		                uint64_t WARPS_PER_BLK = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
		                // Local threadId, i.e. within a single block
		                uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
		                // Local warpId, i.e. within a single block
		                uint64_t wid = tid / WARP_SIZE;
		                // BlockId
		                uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
		                // Global warpId i.e. across all blocks
		                uint64_t g_wid = wid + bid * WARPS_PER_BLK;
		                uint64_t g_tid = (g_wid << 5) | (tid & ((ONE << 5) - ONE));
		                mem_access_t ma;
		                ma.addr = addr + internalOffset;
		                
		                ma.warp_id   = g_tid;
		                ma.opcode_id = opcode_id;
		                ma.read_md   = race_read_md;
		                ma.write_md  = race_write_md;
		                ma.reason    = detected_race;
		                ma.extra     = extra;
		                
		                ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;
		                channel_dev->push(&ma, sizeof(mem_access_t));
		            }
		        }
		    }
			internalOffset += (uint64_t)((uint32_t*)parameters)[BYTE_GRAN];
		} while(internalOffset < dataSize);
    }
    __syncwarp(mask);
}
